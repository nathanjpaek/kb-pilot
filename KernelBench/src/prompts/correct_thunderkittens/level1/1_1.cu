#include "hip/hip_runtime.h"
// ThunderKittens warp-level fp16 matmul for H100 (half inputs/outputs)
#include "kittens.cuh"
#include "pyutils/pyutils.cuh"

#define TILE_SIZE 64
#define NUM_WARPS 4
#define NUM_THREADS (NUM_WARPS * kittens::WARP_THREADS)

struct micro_globals {
    using sub_tile = kittens::st_hf<TILE_SIZE, TILE_SIZE>;
    using tile_gl = kittens::gl<kittens::half, 1, 1, -1, -1, sub_tile>;
    
    tile_gl A, B, C;
    int M, K, N;
    
    dim3 grid() const { return dim3((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE); }
    dim3 block() const { return dim3(NUM_THREADS, 1, 1); }
    size_t dynamic_shared_memory() const { return 100000; }
};

__global__ __launch_bounds__(NUM_THREADS, 1)
void micro_tk(const __grid_constant__ micro_globals g) {
    extern __shared__ kittens::alignment_dummy __shm[];
    kittens::shared_allocator al((int*)&__shm[0]);
    
    kittens::st_hf<TILE_SIZE, TILE_SIZE> (&As)[2] = al.allocate<kittens::st_hf<TILE_SIZE, TILE_SIZE>, 2>();
    kittens::st_hf<TILE_SIZE, TILE_SIZE> (&Bs)[2] = al.allocate<kittens::st_hf<TILE_SIZE, TILE_SIZE>, 2>();
    kittens::st_hf<TILE_SIZE, TILE_SIZE> &C_tile  = al.allocate<kittens::st_hf<TILE_SIZE, TILE_SIZE>>();
    
    int tic = 0, toc = 1;
    int row = blockIdx.y;
    int col = blockIdx.x;
    
    kittens::rt_fl<16, TILE_SIZE> C_accum;
    kittens::warpgroup::zero(C_accum);
    
    __shared__ kittens::semaphore bar;
    if (threadIdx.x == 0) {
        kittens::init_semaphore(bar, 0, 1);
        kittens::tma::expect_bytes(bar, kittens::size_bytes<typeof(As[0])> + kittens::size_bytes<typeof(Bs[0])>);
        kittens::tma::load_async(As[tic], g.A, {0, 0, row, 0}, bar);
        kittens::tma::load_async(Bs[tic], g.B, {0, 0, 0, col}, bar);
    }
    __syncthreads();
    
    int num_tiles = (g.K + TILE_SIZE - 1) / TILE_SIZE;
    for (int tile = 0; tile < num_tiles; ++tile, tic ^= 1, toc ^= 1) {
        kittens::wait(bar, tic);
        __syncthreads();
        
        if (threadIdx.x == 0 && tile + 1 < num_tiles) {
            kittens::tma::expect_bytes(bar, kittens::size_bytes<typeof(As[0])> + kittens::size_bytes<typeof(Bs[0])>);
            kittens::tma::load_async(As[toc], g.A, {0, 0, row, tile + 1}, bar);
            kittens::tma::load_async(Bs[toc], g.B, {0, 0, tile + 1, col}, bar);
        }
        
        kittens::warpgroup::mma_AB(C_accum, As[tic], Bs[tic]);
        kittens::warpgroup::mma_async_wait();
        __syncthreads();
    }
    
    kittens::warpgroup::store(C_tile, C_accum);
    kittens::warpgroup::sync(1);
    
    if (kittens::warpid() == 0) {
        kittens::tma::store_async(g.C, C_tile, {0, 0, row, col});
        kittens::tma::store_async_read_wait();
    }
}

void dispatch_micro(micro_globals g) {
    size_t smem = g.dynamic_shared_memory();
    hipFuncSetAttribute(reinterpret_cast<const void*>(micro_tk), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
    micro_tk<<<g.grid(), g.block(), smem>>>(g);
    hipDeviceSynchronize();
}

PYBIND11_MODULE(tk_kernels, m) {
    kittens::py::bind_kernel<micro_tk, micro_globals>(
        m, "micro_tk",
        &micro_globals::A, &micro_globals::B, &micro_globals::C,
        &micro_globals::M, &micro_globals::K, &micro_globals::N
    );
    kittens::py::bind_function<dispatch_micro, micro_globals>(
        m, "dispatch_micro",
        &micro_globals::A, &micro_globals::B, &micro_globals::C,
        &micro_globals::M, &micro_globals::K, &micro_globals::N
    );
}
